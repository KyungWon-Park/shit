#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <string.h>
#include <math.h>
#include <assert.h>
#include "parser.h"

#define BATCH_SIZE 32 
/* BATCH_SIZE 
   *
   * Why use BATCH_SIZE ?
   *
   * 0. Saturate Streaming Multiprocessors with enough computaion BLOCKS
   * 1. Saturate Video RAM with enough computaional jobs
   * 
   * CRITERIA:
   * 	- Deploy enough blocks (More than n * SM counts) for latency hiding
   * 	- Saturate each block with enough threads 
   */

/* 				NVIDIA GEFORCE GTX1080
   * GPU SPEC:
   * 	- warp_size: 32 threads
   * 	- word_size: 4 Bytes
   * 	- SM_count: 20 Streaming Multiprocessors
   * 	
   * SM SPEC: 
   * 	- max_warps: 64
   * 	- max_thread_blocks : 32 
   * 	- max_threads: 2048
   * 	- max_registers: 65536 words
   * 	- CUDA_cores: 64 cores 
   * 	- share_memory: 64 kB
   *
   * BLOCK SPEC:
   * 	- max_threads: 1024
   * 	- max_registers: 65536 words
   *
   * THREAD SPEC:
   * 	- max_registers: 255 words 
   *
   */

/* Memory design 
 * 
 * 0. INPUT image data 
 * 	=> ALL goes into global memory
 *
 * 1. Filter map data 
 * 	=> Put as much as we can into constant memory (d_map), but leftover should go to global memory (d_map_spill)
 *
 * 2. Result data 
 * 	=> Should go to global memory since write-once
 *
 * 3. What to cache into shared memory?
 * 	=> Bring Filter map data into shared_memory (only necessary part)
 * 	=> Bring INPUT data into shared_memory (only necessary part)
 *
 */

__constant__ int D_BATCH_SIZE;
__constant__ int D_NUM_TEST;
__constant__ __gpu_map__ d_map;

__device__ float sigmoid(float x)
{
	return (1 / (1 + exp(-x)));
}

/*
   * ARGUMENTS:
   * 	- curr_step: Which step are we in? (In MAIN_LOOP)
   * 	- stage: Stage number(ex; 1 means C1 layer, 3 means C3 layer)
   * 	- num_output: Number of output maps 
   * 	- num_input: Number of input maps 
   * 	- height_input: Height of input maps 
   * 	- width_input: Width of input maps 
   * 	- size_filter: Size of filter map, 5 for LeNet-5
   * 	- d_map + d_map_spill: Contains filter maps for all layers
   * 	- inputs: Source of input images 
   * 	- outputs: Destination to store output(computed) images
   * 	- size_input: Length of input 1D array, for fully connected layer
   * 	- size_output: Length of output 1D array, for fully connected layer
   */

__global__ void 	// Convolution computation kernel  
convolution_kernel(
	int curr_step, int stage,
	int num_output, int num_input, int height_input, int width_input,
	int size_filter, __gpu_map__ *d_map,
	float *inputs, float *outputs
)
{
	int BID_x = blockIdx.x; 	// foreach: output image 	~6 or ~16 
	int BID_y = blockIdx.y; 	// foreach: BATCH among curr_step_inputs[BATCH_SIZE] 
	int TID_x = threadIdx.x; 	// foreach: output image row 	~28 or ~10 
	int TID_y = threadIdx.y; 	// foreach: output image column 	~28 or ~10

	float acc = 0;
	if (stage == 1)
	{// C1_layer convolution: D_BATCH_SIZE * { [1 @ 32 * 32] .X [6 * 1 @ 5 * 5] => [6 @ 28 * 28] }
		// Get the starting point from entire MNIST data set 
		float *input_start = inputs + (curr_step * D_BATCH_SIZE * (32 * 32)) + (BID_y * 32 * 32);

		// Load data into shared memory
		__shared__ float input[32][32];
		for (int i = 0; i < 2; i++)
		{
			int rp = 28 * i + TID_x;
			int cp = 28 * i + TID_y;
			if (rp < height_input && cp < width_input)
			{
				input[rp][cp] = input_start[(32 * rp) + cp];
			}
		}
		__syncthreads();
		__shared__ float filter[5][5];
		if (TID_x < size_filter && TID_y < size_filter) 
		{
			filter[TID_x][TID_y] = (*d_map).C1_param[BID_x][0][TID_x][TID_y]; 
		}
		__syncthreads();

		for (int f_row = 0; f_row < size_filter; f_row++)
		{
			for (int f_col = 0; f_col < size_filter; f_col++)
			{
				acc += input[TID_x + f_row][TID_y + f_col] * filter[f_row][f_col];
			}
		}
		outputs[(BID_y * 6 * 28 * 28) + (BID_x * 28 * 28) + (TID_x * 28) + TID_y] = acc; 
	}
	else // Desired stage = 3
	{// C3_layer convolution: D_BATCH_SIZE * { [6 @ 14 * 14] .X [16 * 6 @ 5 * 5] => [16 @ 10 * 10] }
		// Get the starting point from d_s2_results[BATCH_SIZE]
		float *input_start = inputs + (BID_y * (14 * 14));
		
		for (int c = 0; c < num_input; c++)
		{
			// Load data into shared memory 
			__shared__ float input[14][14];
			for (int i = 0; i < 2; i++)
			{
				int rp = 14 * i + TID_x;
				int cp = 14 * i + TID_y;
				if (rp < height_input && rp < width_input)
				{
					input[rp][cp] = input_start[(32 * rp) + cp];
				}
			}
			__syncthreads();
			__shared__ float filter[5][5];
			if (TID_x < size_filter && TID_y < size_filter)
			{
				filter[TID_x][TID_y] = (*d_map).C3_param[BID_x][c][TID_x][TID_y];
			}
			__syncthreads();

			for (int f_row = 0; f_row < size_filter; f_row++)
			{
				for (int f_col = 0; f_col < size_filter; f_col++)
				{
					acc += input[TID_x + f_row][TID_y + f_col] * filter[f_row][f_col];
				}
			}
		}
		outputs[(BID_y * 16 * 10 * 10) + (BID_x * 10 * 10) + (TID_x * 10) + TID_y];
	}

	return;
}

__global__ void 	// Pooling computation kernel
pooling_kernel(
	int curr_step, int stage,
	int num_output, int height_input, int width_input,
	__gpu_map__ *d_map,
	float *inputs, float *outputs 
)
{
	if (stage == 2)
	{// S2_layer pooling: D_BATCH_SIZE * { Sigmoid([6 @ 28 * 28] + bias[6]) => [6 @ 14 * 14] }
	}
	else // Desired stage = 4
	{// S4_layer pooling: D_BATCH_SIZE * { Sigmoid([16 @ 10 * 10] + bias[16]) => [16 @ 5 * 5] }
	}
	return;
}

__global__ void 	// Fully connecting computation kernel 
fullyConnect_kernel(
	int curr_step, int stage,
	int size_input, int size_output,
	__gpu_map__ *d_map, __gpu_map_spill__ *d_map_spill,
	float *inputs, float *outputs 
)
{
	if (stage == 5)
	{// F5_layer full connection: D_BATCH_SIZE * { Sigmoid([120 * 400] X Serial[16 @ 5 * 5] + bias[120 * 1]) => [120 * 1] }
	}
	else // Desired stage = 6
	{// F6_layer full connection: D_BATCH_SIZE * { Sigmoid([84 * 120] X [120 * 1] + bias[84 * 1]) => [84 * 1] }
	}
	return;
}

__global__ void 	// Output layer compuation kernel 
output_kernel(
	int curr_step, int stage,
	int size_input, int size_output,
	__gpu_map__ *d_map, __gpu_map_spill__ *d_map_spill,
	float *inputs, float *outputs
)
{
	// OUTPUT_layer: D_BATCH_SIZE * { [10 * 84] X [84 * 1] + [10 * 1] => [10 * 1] }
	return;
}

__global__ void 	// Number determination kernel 
numberDetermine_kernel(
	int curr_step, int stage,
	float *inputs, int *outputs
)
{
	// NUMBER_layer: D_BATCH_SIZE * { ReduceMax[10 * 1] => SINGLE_DIGIT }
	return;
}

void forward_GPU(float **ptr_test_data, int **ptr_test_label, __map__ *map, int *cnt_correct)
{// Deploy forward computation job on GPU
	float *test_data = *ptr_test_data;
	int *test_label = *ptr_test_label;

	// Acquire memory space in GPU 
	// Prefix "d_" means ADDRESS in device memory 
	// Handlers for device memory manipulation
	int inferences[BATCH_SIZE];
	int *d_inferences;

	float *d_test_data;
	__gpu_map_spill__ *d_map_spill;

	float *d_c1_results;
	float *d_s2_results;
	float *d_c3_results;
	float *d_s4_results;
	float *d_f5_results;
	float *d_f6_results;
	float *d_output_results;

	// WARNING: MALLOC 1
	__gpu_map__ *tmp_map = malloc(sizeof(__gpu_map__));
	__gpu_map_spill__ *tmp_map_spill = malloc(sizeof(__gpu_map_spill__));
	assert(tmp_map != NULL && "MALLOC FAILED!\n");
	assert(tmp_map_spill != NULL && "MALLOC FAILED!\n");

	// Fill in gpu_map data
	// tmp_map = map - F5_param 
	memcpy((*tmp_map).C1_param, (*map).C1_param, sizeof(float) * 6 * 1 * 5 * 5);
	memcpy((*tmp_map).C1_bias, (*map).C1_bias, sizeof(float) * 6);
	memcpy((*tmp_map).C3_param, (*map).C3_param, sizeof(float) * 16 * 6 * 5 * 5);
	memcpy((*tmp_map).C3_bias, (*map).C3_bias, sizeof(float) * 16);
	memcpy((*tmp_map).F5_bias, (*map).F5_bias, sizeof(float) * 120);
	memcpy((*tmp_map).F6_param, (*map).F6_param, sizeof(float) * 84 * 120);
	memcpy((*tmp_map).F6_bias, (*map).F6_bias, sizeof(float) * 84);
	memcpy((*tmp_map).OUTPUT_param, (*map).OUTPUT_param, sizeof(float) * 10 * 84);
	memcpy((*tmp_map).OUTPUT_bias, (*map).OUTPUT_bias, sizeof(float) * 10);

	// tmp_map_spill = F5 param
	memcpy((*tmp_map_spill).F5_param, (*map).F5_param, sizeof(float) * 120 * 400);

	// Fix NUM_TEST into d_NUM_TEST so d_NUM_TEST can be multiple of BATCH_SIZE, so we can walk in stride
	int d_NUM_TEST = ((int) ceil((double) ((float) NUM_TEST / (float) BATCH_SIZE))) * BATCH_SIZE;
	int batch_size = BATCH_SIZE;

	// WARNING: MALLOC 0
	hipMalloc((void **) &d_inferences, sizeof(int) * BATCH_SIZE);
	hipMalloc((void **) &d_test_data, sizeof(float) * d_NUM_TEST * 32 * 32);
	hipMalloc((void **) &d_map_spill, sizeof(__gpu_map_spill__));
	hipMalloc((void **) &d_c1_results, sizeof(float) * BATCH_SIZE * 6 * 28 * 28);
	hipMalloc((void **) &d_s2_results, sizeof(float) * BATCH_SIZE * 6 * 14 * 14);
	hipMalloc((void **) &d_c3_results, sizeof(float) * BATCH_SIZE * 16 * 10 * 10);
	hipMalloc((void **) &d_s4_results, sizeof(float) * BATCH_SIZE * 16 * 5 * 5);
	hipMalloc((void **) &d_f5_results, sizeof(float) * BATCH_SIZE * 120);
	hipMalloc((void **) &d_f6_results, sizeof(float) * BATCH_SIZE * 84);
	hipMalloc((void **) &d_output_results, sizeof(float) * BATCH_SIZE * 10);

	// CUDA memcpy from host to device 
	hipMemcpyToSymbol(HIP_SYMBOL(D_NUM_TEST), &d_NUM_TEST, sizeof(int), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(D_BATCH_SIZE), &batch_size, sizeof(int), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_map), tmp_map, sizeof(__gpu_map__), 0, hipMemcpyHostToDevice);
	hipMemcpy(d_map_spill, tmp_map_spill, sizeof(__gpu_map_spill__), 0, hipMemcpyHostToDevice);

	// WARNING: FREE 1
	free(tmp_map);
	free(tmp_map_spill);

	// ENTERING MAIN LOOP
	int step = 0;
	dim3 block;
	dim3 thread;
	for (int step = 0; (step * BATCH_SIZE) < d_NUM_TEST; step++)
	{// Advance step by step, with BATCH_SIZE stride 
		// START
		// 0. Convolution layer C1

		// 1. Pooling layer S2 

		// 2. Convolution layer C3

		// 3. Pooling layer S4

		// 4. Fully connected layer F5

		// 5. Fully connected layer F6

		// 6. Output layer OUTPUT

		// 7. Determine number 

		// 8. Update cnt_correct
		hipMemcpy(inferences, d_inferences, sizeof(int) * BATCH_SIZE, hipMemcpyDeviceToHost);
		for (int i = 0; i < BATCH_SIZE; i++)
		{// For every result numbers in BATCH
			int index = (step * BATCH_SIZE) + i;
			if (index >= NUM_TEST)
			{// Check that our BATCH didn't go out of NUM_TEST 
				break;
			}
			else 
			{// If this inferences[i] is valid result, 
				if (inferences[i] == test_label[index])
				{// If such inferences[i] is same with test_label[index], increment cnt_correct counter
					(*cnt_correct)++;
				}
			}
		}
	}

	// WARNING: FREE 0
	hipFree(d_inferences);
	hipFree(d_map_spill);
	hipFree(d_test_data);
	hipFree(d_c1_results);
	hipFree(d_s2_results);
	hipFree(d_c3_results);
	hipFree(d_s4_results);
	hipFree(d_f5_results);
	hipFree(d_f6_results);
	hipFree(d_output_results);
	
	return;
}
